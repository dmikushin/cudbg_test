#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "cudadebugger.h"

#include <iostream>
#include <pthread.h>
#include <signal.h>

// helpers

CUDBGAPI cudbgAPI;

void exit_safely(int code) {
  cudbgAPI->finalize();
  exit(code);
}

#define cudbgCheck(ans)                                                        \
  { __cudbgCheck((ans), __FILE__, __LINE__); }
inline void __cudbgCheck(CUDBGResult res, const char *file, int line) {
  if (res != CUDBG_SUCCESS) {
    std::cerr << "ERROR: " << cudbgGetErrorString(res) << " at " << file << ":"
              << line << std::endl;
    exit_safely(EXIT_FAILURE);
  }
}

// event handling

pthread_cond_t event_cond = PTHREAD_COND_INITIALIZER;
pthread_mutex_t event_lock = PTHREAD_MUTEX_INITIALIZER;

void event_callback(CUDBGEventCallbackData *data) {
  std::cout << "Event notification" << std::endl;
  pthread_cond_signal(&event_cond);
}

void handle_events() {
  while (true) {
    CUDBGEvent event;
    CUDBGResult res = cudbgAPI->getNextEvent(CUDBG_EVENT_QUEUE_TYPE_SYNC, &event);
    if (res == CUDBG_ERROR_NO_EVENT_AVAILABLE) {
      break;
    } else if (res != CUDBG_SUCCESS) {
      std::cerr << "HANDLER ERROR: " << cudbgGetErrorString(res) << std::endl;
      break;
    }

    std::cout << "Event: ";
    switch (event.kind) {
    case CUDBG_EVENT_INVALID:
      std::cout << "CUDBG_EVENT_INVALID";
      break;
    case CUDBG_EVENT_ELF_IMAGE_LOADED:
      std::cout << "CUDBG_EVENT_ELF_IMAGE_LOADED";
      break;
    case CUDBG_EVENT_KERNEL_READY:
      std::cout << "CUDBG_EVENT_KERNEL_READY";
      break;
    case CUDBG_EVENT_KERNEL_FINISHED:
      std::cout << "CUDBG_EVENT_KERNEL_FINISHED";
      break;
    case CUDBG_EVENT_INTERNAL_ERROR:
      std::cout << "CUDBG_EVENT_INTERNAL_ERROR ("
                << cudbgGetErrorString(event.cases.internalError.errorType)
                << ")";
      break;
    case CUDBG_EVENT_CTX_PUSH:
      std::cout << "CUDBG_EVENT_CTX_PUSH";
      break;
    case CUDBG_EVENT_CTX_POP:
      std::cout << "CUDBG_EVENT_CTX_POP";
      break;
    case CUDBG_EVENT_CTX_CREATE:
      std::cout << "CUDBG_EVENT_CTX_CREATE";
      break;
    case CUDBG_EVENT_CTX_DESTROY:
      std::cout << "CUDBG_EVENT_CTX_DESTROY";
      break;
    case CUDBG_EVENT_TIMEOUT:
      std::cout << "CUDBG_EVENT_TIMEOUT";
      break;
    case CUDBG_EVENT_ATTACH_COMPLETE:
      std::cout << "CUDBG_EVENT_ATTACH_COMPLETE";
      break;
    case CUDBG_EVENT_DETACH_COMPLETE:
      std::cout << "CUDBG_EVENT_DETACH_COMPLETE";
      break;
    case CUDBG_EVENT_ELF_IMAGE_UNLOADED:
      std::cout << "CUDBG_EVENT_ELF_IMAGE_UNLOADED";
      break;
    default:
      std::cout << "unknown event";
      break;
    }
    std::cout << std::endl;
  }

  // TODO: we should probably acknowledge the sync events here;
  //       I think that's why I'm getting the timeout events.
}

void *event_handler(void *null) {
  while (true) {
    pthread_mutex_lock(&event_lock);
    pthread_cond_wait(&event_cond, &event_lock);
    handle_events();
    pthread_mutex_unlock(&event_lock);
  }
}

// main

__global__ void kernel() { printf("Hello, World!\n"); }

int main(int argc, char const *argv[]) {
  signal(SIGINT, exit_safely);

  // gets the api
  std::cout << "Initializing debug API" << std::endl;
  uint32_t major, minor, rev;
  cudbgCheck(cudbgGetAPIVersion(&major, &minor, &rev));
  cudbgCheck(cudbgGetAPI(major, minor, rev, &cudbgAPI));
  cudbgCheck(cudbgAPI->initialize());

  // starts thread to print out events
  std::cout << "Starting event handler" << std::endl;
  pthread_t mannage_event_thread;
  pthread_create(&mannage_event_thread, NULL, event_handler, NULL);
  cudbgCheck(cudbgAPI->setNotifyNewEventCallback(event_callback));

  // Causes the program to freeze
  std::cout << "Launching kernel" << std::endl;
  kernel<<<1, 1>>>();

  exit_safely(0);

  return 0;
}

// this example does not work for unknown reasons, resulting in an "internal error (invalid
// context)" event that really does not make any sense.
//
// the subsequent timeouts are due to not acknowledging the sync events.

